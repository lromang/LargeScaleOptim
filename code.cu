#include "hip/hip_runtime.h"
/*
 * Luis Manuel Román García
 *
 * ----------------------------------
 * Rutinas de propósito general para
 * optimización numérica. CUDA
 * ----------------------------------
 */

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

__global__ double* vSum(double* v, double* u, double* w, int length);

int main(){
  // Declaración de variables.
  double *d_v, *d_u, *d_w, *h_v, *h_u, *h_w;
  int i, size;

  // Lectura de longitud.
  printf("Escribir tamaño de vectores:\n");
  scanf("%d", &length);

  // Alocar espacio en host.
  h_v = (double*)malloc(length*sizeof(double));
  h_u = (double*)malloc(length*sizeof(double));

  // Alocar memoria en device.
  hipError_t err0 = hipMalloc((void**) &d_v, size);
  hipError_t err1 = hipMalloc((void**) &d_u, size);
  hipError_t err2 = hipMalloc((void**) &d_w, size);

  // Enviar argumentos a device.
  hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
  hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);

  // Alocar espacio en device.
  hipMalloc((void **) &v, length*sizeof(double));
  hipMalloc((void **) &u, length*sizeof(double));
  hipMalloc((void **) &w, length*sizeof(double));

  // Bloques y threads a levantar.
  dim3 DimGrid((n - 1)/256 + 1, 1, 1);
  dim3 DimBlock(256, 1, 1);

  //Ejecutar kernel.
  vSum<<<DimGrid, DimBlock>>>(d_u, d_v, d_w, n);

  // Regresar resultados.
  hipMemcpy(h_w, d_w, size, hipMemcpyDeviceToHost);

  // Liberar Memoria.
  hipFree(d_v);
  hipFree(d_u);
  hipFree(d_w);

  // Verificar si hubo errores.
  if((err0 != hipSuccess) || (err1 != hipSuccess) || (err2 != hipSuccess)){
    printf("%s en %s en línea %d \n",
           hipGetErrorString(err),
           __FILE__,
           __LINE__);

    exit(EXIT_FAILURE);
  }


}

__global__ void vSum(double *v, double *u, double *w, int length){
  // Declaración de variables.
  int i, size;

  // Inicializar variables.
  size = length*sizeof(double);
  i    = blockIdx.x * blockDim.x + threadIdx.x;

  // Suma. Verificar que el índice es válido.
  if(i < length){
    w[i] = v[i] + u[i];
  }
}
